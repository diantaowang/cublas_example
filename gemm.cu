#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <hip/hip_runtime.h>

#include <hipblas.h>

using namespace std;
#define IDX2C(i, j, ld) (((i) * (ld)) + (j))

void printPlainMatrix(const float* matrix, const int H, const int W)
{
  for (int i = 0; i < H; i++)
    {
        for (int j = 0; j < W; j++)
        {
          std::cout << std::fixed << std::setw(8) << std::setprecision(4) << matrix[IDX2C(i, j, W)];
        }
        std::cout << std::endl;
    }
}

int main()
{
    hipblasHandle_t handle;

    // Prepare input matrices
    float *A, *B, *C;
    int M, N, K;
    float alpha, beta;

    M = 2;
    N = 3;
    K = 3;
    alpha = 1.f;
    beta = 0.f;

    // create cuBLAS handle
    if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS initialization failed" << std::endl;
        return EXIT_FAILURE;
    }

    hipMallocManaged((void**)&A, sizeof(float) * M * K);
    for (int i = 0; i < M; i++) {
      for (int j = 0; j < K; j++) {
        A[IDX2C(i, j, K)] = i + 1;
      }
    }

    /*
      A:
      1.0000  1.0000  1.0000
      2.0000  2.0000  2.0000
    */
    std::cout << "A:" << std::endl;
    printPlainMatrix(A, M, K);


    hipMallocManaged((void**)&B, sizeof(float) * K * N);
    for (int i = 0; i < K; i++) {
      for (int j = 0; j < N; j++) {
        B[IDX2C(i, j, N)] = i + 1;
      }
    }
    /*
      B:
      1.0000  1.0000  1.0000
      2.0000  2.0000  2.0000
      3.0000  3.0000  3.0000
    */
    std::cout << "B:" << std::endl;
    printPlainMatrix(B, K, N);

    hipMallocManaged((void**)&C, sizeof(float) * M * N);
    for (int i = 0 ; i < M; i++) {
      for (int j = 0; j < N; j++) {
        C[IDX2C(i, j, N) ] = 1;
      }
    }

    std::cout << "C:" << std::endl;
    printPlainMatrix(C, M, N);

    // Gemm
    hipblasSgemm(handle,
                HIPBLAS_OP_T,// 通过转置变为逻辑上的matrix
                HIPBLAS_OP_T, // 同上
                M, // 逻辑上A和C的row的个数
                N, // 逻辑上B和C的column的个数
                K, // 逻辑上A的column个数和B的row 的个数
                &alpha,
                A,
                K, //由于使用了转置，这里填写A的列数
                B,
                N, // 由于使用了转置, 这里填写B的列数
                &beta,
                C,
                M);

    hipDeviceSynchronize();
    /*
      C out:
       6.0000 12.0000  6.0000
      12.0000  6.0000 12.0000
      C host memory layout:
      6.0000  12.0000 6.0000  12.0000 6.0000  12.0000
    */
    std::cout << "C out:" << std::endl;

    printPlainMatrix(C, M, N);

    std::cout << "C host memory layout:" << endl;;
    for (int i = 0; i < M *N ; ++i ) {
      cout << C[i] << "\t";
    }
    cout << endl;


    hipblasDestroy(handle);

    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}